#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>
// global, device, host是用于cuda c/c++代码中的限定符， 用于定义函数或变量的执行位置和内存类型，
//  global 用于定义在gpu上执行的函数，被称为内核函数，它们可以由主机cpu调用，并在GPU上的多个线程中并行执行，
// 内核函数通常用于对GPU数据进行计算
//  device这个限定符用于定义只能在GPU上执行并且只能从GPU代码访问的函数或变量，device的函数可以从其他设备代码，如
// 设备函数或内核函数调用，设备变量存储在GPU的内存中，并可以被GPU线程访问
// host 用于定义只能在CPU上执行，并且只能从主机代码访问的函数或变量，host的函数可以从主机CPU代码调用，主机变量存储
// 在主机的内存中，并可以被CPU代码访问
// 这些限定符允许开发者指定函数或变量的存放位置和访问方式，通过适当地使用这些限定符号，您可以控制代码的执行位置和内存类型
// 
namespace TransFTrain {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  // dim3 是一个用于描述线程块和网格维度的数据类型， 它是cuda_runtime api中定义的一个结构题
  // dim3 结构体包含三个无符号整数成员变量 x,y,z 分别表示三个维度的大小，这三个维度可以用来描述线程块block和网格grid的结构
  // 在CUDA程序中个，线程块和网格的概念是用于并行执行代码的，线程块是一组并发执行的现场，而网格则是由多个线程块组成的集合
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  uint32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<uint32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

__device__ uint32_t GetOffset(uint32_t gid, const CudaVec& shape, const CudaVec& strides, uint32_t initial){
    size_t idx = initial;

    for(int i = shape.size -1; i>=0; --i){
      idx += strides.data[i] * (gid % shape.data[i]);
      gid /= shape.data[i];
    }

    return idx;
}

__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  ssize_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if(gid < size)
    out[gid] = a[GetOffset(gid, shape, strides, offset)];
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
             std::vector<uint32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}

__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape, CudaVec strides, size_t offset) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    out[GetOffset((gid), shape, strides, offset)] = a[gid];
  }
}


void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<uint32_t> shape,
                  std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  CudaDims dim = CudaOneDim(a.size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape), VecToCuda(strides), offset);
}

__global__ void ScalarSetitemKernel(scalar_t val, scalar_t* out, size_t size, CudaVec shape, CudaVec strides, size_t offset){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    out[GetOffset((gid), shape, strides, offset)] = val;
  }
}


void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<uint32_t> shape,
                   std::vector<uint32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  CudaDims dim = CudaOneDim(size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape), VecToCuda(strides), offset);
}


__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA array
   */
  CudaDims dim = CudaOneDim(out->size);
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add together a CUDA array and a scalar value.
   */
  CudaDims dim = CudaOneDim(out->size);
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size)
    out[gid] = a[gid] * b[gid];
}

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out){
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}


__global__ void  ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    out[gid] = a[gid] * val;
  }
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out){
  CudaDims dim = CudaOneDim(out->size);
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size){
  size_t gid = blockIdx.x*blockDim.x + threadIdx.x;
  if(gid < size){
    out[gid] = a[gid] / b[gid];
  }
}

void EwiseDiv(const CudaArray& a, CudaArary& b, CudaArray * out){
  CudaDims dim = CudaOneDim(out->size);
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivKernel(const scalar_t * a, const scalar_t val, const scalar_t out, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size){
    out[gid] = a[gid] / val;
  }
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out){
  CudaDims dim = CudaOneDim(out->size);
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t * out, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    out[gid] = (a[gid] < b[gid]) ? b[gid] : a[gid];
  }
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out){
  CudaDims dim = CudaOneDim(out->size);
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, size_t val, scalar_t * out, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid< size){
    out[gid] = 
  }
}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN YOUR SOLUTION
  
  /// END YOUR SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////



void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  
  /// END YOUR SOLUTION
}




void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN YOUR SOLUTION
  
  /// END YOUR SOLUTION
}

}  // namespace cuda
}  // namespace TransFTrain

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace TransFTrain;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  // m.def("ewise_mul", EwiseMul);
  // m.def("scalar_mul", ScalarMul);
  // m.def("ewise_div", EwiseDiv);
  // m.def("scalar_div", ScalarDiv);
  // m.def("scalar_power", ScalarPower);

  // m.def("ewise_maximum", EwiseMaximum);
  // m.def("scalar_maximum", ScalarMaximum);
  // m.def("ewise_eq", EwiseEq);
  // m.def("scalar_eq", ScalarEq);
  // m.def("ewise_ge", EwiseGe);
  // m.def("scalar_ge", ScalarGe);

  // m.def("ewise_log", EwiseLog);
  // m.def("ewise_exp", EwiseExp);
  // m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
